#include "hip/hip_runtime.h"
#include "vrender_cuda_kernel.cuh"

extern "C"
void initializeVRender( float           *data,
                        int3            size,
                        float           max,
                        float           min,
                        hipExtent      volumeSize,
                        uint            imageW,
                        uint            imageH )
{
    size_t float_size = size.x * size.y * size.z * sizeof(float);

    float *ddata;
    checkCudaErrors( hipMalloc( (void**) &ddata, float_size ) );
    checkCudaErrors( hipMemset( ddata, 0, float_size ) );

    dim3 block(size.x);
    dim3 grid(size.y,size.z);
    size_t data_size = size.x * size.y * size.z * sizeof(unsigned char);

    checkCudaErrors( hipMalloc( (void**) &d_charvol, data_size ) );
    checkCudaErrors( hipMemset( d_charvol, 0, data_size ) );

    checkCudaErrors( hipMemcpy( ddata, data, float_size, hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( dmax), &max, sizeof(float) ) );
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( dmin), &min, sizeof(float) ) );

    deviceFloat2Char<<<grid,block>>>( ddata, d_charvol );
    hipDeviceSynchronize();

    checkCudaErrors( hipFree(ddata) );


    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();
    checkCudaErrors( hipMalloc3DArray( &d_volumeArray, &channelDesc, volumeSize ) );

    hipMemcpy3DParms copyParams;
    copyParams.srcPtr   =   make_hipPitchedPtr( d_charvol, volumeSize.width * sizeof(unsigned char), volumeSize.width, volumeSize.height );
    copyParams.dstArray =   d_volumeArray;
    copyParams.extent   =   volumeSize;
    copyParams.kind     =   hipMemcpyDeviceToDevice;
    checkCudaErrors( hipMemcpy3D( &copyParams ) );

    tex.normalized = true;
    tex.filterMode = hipFilterModeLinear;
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;

    checkCudaErrors( hipBindTextureToArray( tex, d_volumeArray, channelDesc ) );

    float4 transferFunc[] = {
        { 0, 0, 0, 0 },
        { 0.2, 0.2, 0.2, 1 },
        { 0.4, 0.4, 0.4, 1 },
        { 0.6, 0.6, 0.6, 1 },
        { 0.8, 0.8, 0.8, 1 },
        { 1, 1, 1, 1 }
    };

    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
    hipArray *d_transferFuncArray;

    checkCudaErrors( hipMallocArray( &d_transferFuncArray, &channelDesc2, sizeof(transferFunc) / sizeof(float4), 1 ) );
    checkCudaErrors( hipMemcpyToArray( d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice ) );

    transferTex.filterMode = hipFilterModeLinear;
    transferTex.normalized = true;
    transferTex.addressMode[0] = hipAddressModeClamp;

    checkCudaErrors( hipBindTextureToArray( transferTex, d_transferFuncArray, channelDesc2 ) );

    checkCudaErrors( hipMalloc( (void**) &d_volume, imageW * imageH * 3 * sizeof(unsigned char) ) );
}


extern "C"
void freeCudaBuffers()
{
    checkCudaErrors( hipUnbindTexture(tex) );
    checkCudaErrors( hipFreeArray(d_volumeArray) );

    checkCudaErrors( hipUnbindTexture(transferTex) );
    checkCudaErrors( hipFreeArray(d_transferFuncArray) );

    checkCudaErrors( hipFree( d_charvol ) );
    checkCudaErrors( hipFree( d_volume ) );
}


extern "C"
void render_kernel( dim3 gridSize, dim3 blockSize,
                    unsigned char *buffer,
                    uint imageW, uint imageH,
                    float dens, float bright, float offset, float scale, float weight )
{
    checkCudaErrors( hipMemset( d_volume, 0, imageW * imageH * 3 * sizeof(unsigned char) ) );
    d_render<<<gridSize,blockSize>>>( d_volume,
                                      imageW, imageH,
                                      dens, bright, offset, scale, weight );
    checkCudaErrors( hipMemcpy( buffer, d_volume, imageW * imageH * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost ) );
}


extern "C"
void copyInvViewMatrix( float *invViewMatrix, size_t sizeofMatrix )
{
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( c_invViewMatrix), invViewMatrix, sizeofMatrix, 0, hipMemcpyHostToDevice ) );
}




