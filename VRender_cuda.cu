#include "hip/hip_runtime.h"
#include "VRender_cuda_kernel.cuh"

extern "C"
void initializeVRender( char            *data,
                        uint            size,
                        uint3           *colormap,
                        hipExtent      volumeSize,
                        uint            imageW,
                        uint            imageH )
{
    size_t data_size = size * size * size * sizeof(unsigned char);

    checkCudaErrors( hipMalloc( (void**) &d_charvol, data_size ) );
    checkCudaErrors( hipMemcpy( d_charvol, data, data_size, hipMemcpyHostToDevice ) );

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();
    checkCudaErrors( hipMalloc3DArray( &d_volumeArray, &channelDesc, volumeSize ) );

    hipMemcpy3DParms copyParams;
    copyParams.srcPtr   =   make_hipPitchedPtr( d_charvol, volumeSize.width * sizeof(unsigned char), volumeSize.width, volumeSize.height );
    copyParams.dstArray =   d_volumeArray;
    copyParams.extent   =   volumeSize;
    copyParams.kind     =   hipMemcpyDeviceToDevice;
    checkCudaErrors( hipMemcpy3D( &copyParams ) );

    tex.normalized = true;
    tex.filterMode = hipFilterModeLinear;
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;

    checkCudaErrors( hipBindTextureToArray( tex, d_volumeArray, channelDesc ) );

    float4 transferFunc[] = {
        { 0, 0, 0, 0 },
        { 0.2, 0.2, 0.2, 1 },
        { 0.4, 0.4, 0.4, 1 },
        { 0.6, 0.6, 0.6, 1 },
        { 0.8, 0.8, 0.8, 1 },
        { 1, 1, 1, 1 }
    };

    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
    hipArray *d_transferFuncArray;

    checkCudaErrors( hipMallocArray( &d_transferFuncArray, &channelDesc2, sizeof(transferFunc) / sizeof(float4), 1 ) );
    checkCudaErrors( hipMemcpyToArray( d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice ) );

    transferTex.filterMode = hipFilterModeLinear;
    transferTex.normalized = true;
    transferTex.addressMode[0] = hipAddressModeClamp;

    checkCudaErrors( hipBindTextureToArray( transferTex, d_transferFuncArray, channelDesc2 ) );

    checkCudaErrors( hipMalloc( (void**) &d_volume, imageW * imageH * 3 * sizeof(unsigned char) ) );
}


extern "C"
void freeCudaBuffers()
{
    checkCudaErrors( hipUnbindTexture(tex) );
    checkCudaErrors( hipFreeArray(d_volumeArray) );

    checkCudaErrors( hipUnbindTexture(transferTex) );
    checkCudaErrors( hipFreeArray(d_transferFuncArray) );

    checkCudaErrors( hipFree( d_charvol ) );
    checkCudaErrors( hipFree( d_volume ) );
}


extern "C"
void render_kernel( dim3 gridSize, dim3 blockSize,
                    unsigned char *buffer,
                    uint imageW, uint imageH,
                    float dens, float bright, float offset, float scale, float weight )
{
    checkCudaErrors( hipMemset( d_volume, 0, imageW * imageH * 3 * sizeof(unsigned char) ) );
    d_render<<<gridSize,blockSize>>>( d_volume,
                                      imageW, imageH,
                                      dens, bright, offset, scale, weight );
    checkCudaErrors( hipMemcpy( buffer, d_volume, imageW * imageH * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost ) );
}


extern "C"
void copyInvViewMatrix( float *invViewMatrix, size_t sizeofMatrix )
{
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( c_invViewMatrix), invViewMatrix, sizeofMatrix, 0, hipMemcpyHostToDevice ) );
}




