#include "hip/hip_runtime.h"
#include <algorithm>
#include "VRender_cuda_kernel.cuh"
#include "Cloud.h"

extern "C"
void createVRenderColorMaps( Cloud * cloud )
{
    hipSetDevice(1);

    float cudatime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipDeviceProp_t devProp;
    hipGetDeviceProperties( &devProp, 1 );
    //printf("\n Device: %s", devProp.name );

    checkCudaErrors( hipMalloc( (void**) &d_red, cloud->world_count ) );
    checkCudaErrors( hipMalloc( (void**) &d_green, cloud->world_count ) );
    checkCudaErrors( hipMalloc( (void**) &d_blue, cloud->world_count ) );

    checkCudaErrors( hipMemset( d_red, 0, cloud->world_count ) );
    checkCudaErrors( hipMemset( d_green, 0, cloud->world_count ) );
    checkCudaErrors( hipMemset( d_blue, 0, cloud->world_count ) );

    float3 *h_pos;
    h_pos = new float3[cloud->position.size()];
    std::copy( cloud->position.begin(), cloud->position.end(), h_pos );

    float3 *d_pos;
    checkCudaErrors( hipMalloc( (void**) &d_pos, cloud->position.size() * sizeof(float3) ) );
    checkCudaErrors( hipMemcpy( d_pos, h_pos, cloud->position.size() * sizeof(float3), hipMemcpyHostToDevice ) );

    uint3 *h_color;
    h_color = new uint3[cloud->rgb.size()];
    std::copy( cloud->rgb.begin(), cloud->rgb.end(), h_color );

    uint3 *d_color;
    checkCudaErrors( hipMalloc( (void**) &d_color, cloud->rgb.size() * sizeof(uint3) ) );
    checkCudaErrors( hipMemcpy( d_color, h_color, cloud->rgb.size() * sizeof(uint3), hipMemcpyHostToDevice ) );

    WORLD h_world;
    h_world.npoints = cloud->count;
    h_world.count = cloud->world_count;
    h_world.start.x = cloud->world_start.x;
    h_world.start.y = cloud->world_start.y;
    h_world.start.z = cloud->world_start.z;
    h_world.resolution = cloud->world_res;
    h_world.size = cloud->world_size;

    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( d_world), &h_world, sizeof(WORLD) ) );

    dim3 block(devProp.maxThreadsPerBlock / 4);
    uint sizer = cloud->count;
    int3 tempGridExtent;
    tempGridExtent.x = sizer / block.x;
    tempGridExtent.y = 1;
    tempGridExtent.z = 1;
    if (sizer % block.x > 0) tempGridExtent.x++;
    if (tempGridExtent.x > devProp.maxGridSize[1])
    {
        tempGridExtent.y = tempGridExtent.x / devProp.maxGridSize[1];
        if (tempGridExtent.x % devProp.maxGridSize[1] > 0) tempGridExtent.y++;
        tempGridExtent.x = devProp.maxGridSize[1];
        if (tempGridExtent.y > devProp.maxGridSize[1])
        {
            tempGridExtent.z = tempGridExtent.y / devProp.maxGridSize[1];
            if (tempGridExtent.y % devProp.maxGridSize[1] > 0) tempGridExtent.z++;
            tempGridExtent.y = devProp.maxGridSize[1];
        }
    }
    dim3 grid(tempGridExtent.x,tempGridExtent.y,tempGridExtent.z);

    cuda_create_color_maps<<<grid,block>>> ( d_pos,
                                             d_color,
                                             d_red,
                                             d_green,
                                             d_blue );
    hipDeviceSynchronize();

    checkCudaErrors( hipFree( d_pos ) );
    checkCudaErrors( hipFree( d_color ) );

    delete [] h_pos;
    delete [] h_color;

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &cudatime, start, stop );
    printf("\n ||| TIME - Create Color Maps: %f ms\n", cudatime);
}


extern "C"
void initializeVRender( hipExtent      volumeSize,
                        uint            imageW,
                        uint            imageH )
{
    float cudatime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();

    // RED
    checkCudaErrors( hipMalloc3DArray( &d_redArray, &channelDesc, volumeSize ) );
    redParams.srcPtr   =   make_hipPitchedPtr( d_red, volumeSize.width, volumeSize.width, volumeSize.height );
    redParams.dstArray =   d_redArray;
    redParams.extent   =   volumeSize;
    redParams.kind     =   hipMemcpyDeviceToDevice;
    checkCudaErrors( hipMemcpy3D( &redParams ) );

    texRed.normalized = true;
    texRed.filterMode = hipFilterModeLinear;
    texRed.addressMode[0] = hipAddressModeClamp;
    texRed.addressMode[1] = hipAddressModeClamp;
    texRed.addressMode[2] = hipAddressModeClamp;

    checkCudaErrors( hipBindTextureToArray( texRed, d_redArray, channelDesc ) );

    // GREEN
    checkCudaErrors( hipMalloc3DArray( &d_greenArray, &channelDesc, volumeSize ) );
    greenParams.srcPtr   =   make_hipPitchedPtr( d_green, volumeSize.width, volumeSize.width, volumeSize.height );
    greenParams.dstArray =   d_greenArray;
    greenParams.extent   =   volumeSize;
    greenParams.kind     =   hipMemcpyDeviceToDevice;
    checkCudaErrors( hipMemcpy3D( &greenParams ) );

    texGreen.normalized = true;
    texGreen.filterMode = hipFilterModeLinear;
    texGreen.addressMode[0] = hipAddressModeClamp;
    texGreen.addressMode[1] = hipAddressModeClamp;
    texGreen.addressMode[2] = hipAddressModeClamp;

    checkCudaErrors( hipBindTextureToArray( texGreen, d_greenArray, channelDesc ) );

    // BLUE
    checkCudaErrors( hipMalloc3DArray( &d_blueArray, &channelDesc, volumeSize ) );
    blueParams.srcPtr   =   make_hipPitchedPtr( d_blue, volumeSize.width, volumeSize.width, volumeSize.height );
    blueParams.dstArray =   d_blueArray;
    blueParams.extent   =   volumeSize;
    blueParams.kind     =   hipMemcpyDeviceToDevice;
    checkCudaErrors( hipMemcpy3D( &blueParams ) );

    texBlue.normalized = true;
    texBlue.filterMode = hipFilterModeLinear;
    texBlue.addressMode[0] = hipAddressModeClamp;
    texBlue.addressMode[1] = hipAddressModeClamp;
    texBlue.addressMode[2] = hipAddressModeClamp;

    checkCudaErrors( hipBindTextureToArray( texBlue, d_blueArray, channelDesc ) );

    // OUTPUT BUFFER
    checkCudaErrors( hipMalloc( (void**) &d_volume, imageW * imageH * 3 ) );
    checkCudaErrors( hipMemset( d_volume, 0, imageW * imageH * 3 ) );

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &cudatime, start, stop );
    printf("\n ||| TIME - Initalize GPU Memory: %f ms\n", cudatime);

    size_t freeMem, totalMem;
    checkCudaErrors(hipMemGetInfo(&freeMem,&totalMem));
    printf("\n Free Memory: %lu / %lu\n",freeMem,totalMem);
}


extern "C"
void freeCudaBuffers()
{
    checkCudaErrors( hipUnbindTexture(texRed) );
    checkCudaErrors( hipFreeArray(d_redArray) );

    checkCudaErrors( hipUnbindTexture(texGreen) );
    checkCudaErrors( hipFreeArray(d_greenArray) );

    checkCudaErrors( hipUnbindTexture(texBlue) );
    checkCudaErrors( hipFreeArray(d_blueArray) );

    checkCudaErrors( hipFree( d_red ) );
    checkCudaErrors( hipFree( d_green ) );
    checkCudaErrors( hipFree( d_blue ) );

    checkCudaErrors( hipFree( d_volume ) );
}


extern "C"
void render_kernel( dim3 gridSize, dim3 blockSize,
                    unsigned char *buffer,
                    uint imageW, uint imageH,
                    float dens, float bright, float offset, float scale,
                    float *fps )
{
    float cudatime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    checkCudaErrors( hipMemset( d_volume, 0, imageW * imageH * 3 ) );
    d_render<<<gridSize,blockSize>>>( d_volume,
                                      imageW, imageH,
                                      dens, bright, offset, scale );
    hipDeviceSynchronize();
    checkCudaErrors( hipMemcpy( buffer, d_volume, imageW * imageH * 3, hipMemcpyDeviceToHost ) );

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &cudatime, start, stop );
    *fps = cudatime / 1000.f;
    //printf(" ||| TIME - Render Update: %f ms\n", cudatime);
}


extern "C"
void copyInvViewMatrix( float *invViewMatrix, size_t sizeofMatrix )
{
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( c_invViewMatrix), invViewMatrix, sizeofMatrix, 0, hipMemcpyHostToDevice ) );
}




