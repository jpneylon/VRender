#include "VRender_cuda_kernel.cuh"

extern "C"
void initializeVRender( void  *red_map,
                        void  *green_map,
                        void  *blue_map,
                        hipExtent      volumeSize,
                        uint            imageW,
                        uint            imageH )
{
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();

    // RED
    checkCudaErrors( hipMalloc3DArray( &d_redArray, &channelDesc, volumeSize ) );

    redParams.srcPtr   =   make_hipPitchedPtr( red_map, volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height );
    redParams.dstArray =   d_redArray;
    redParams.extent   =   volumeSize;
    redParams.kind     =   hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D( &redParams ) );

    texRed.normalized = true;
    texRed.filterMode = hipFilterModeLinear;
    texRed.addressMode[0] = hipAddressModeClamp;
    texRed.addressMode[1] = hipAddressModeClamp;
    texRed.addressMode[2] = hipAddressModeClamp;

    checkCudaErrors( hipBindTextureToArray( texRed, d_redArray, channelDesc ) );

    // GREEN
    checkCudaErrors( hipMalloc3DArray( &d_greenArray, &channelDesc, volumeSize ) );

    greenParams.srcPtr   =   make_hipPitchedPtr( green_map, volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height );
    greenParams.dstArray =   d_greenArray;
    greenParams.extent   =   volumeSize;
    greenParams.kind     =   hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D( &greenParams ) );

    texGreen.normalized = true;
    texGreen.filterMode = hipFilterModeLinear;
    texGreen.addressMode[0] = hipAddressModeClamp;
    texGreen.addressMode[1] = hipAddressModeClamp;
    texGreen.addressMode[2] = hipAddressModeClamp;

    checkCudaErrors( hipBindTextureToArray( texGreen, d_greenArray, channelDesc ) );

    // BLUE
    checkCudaErrors( hipMalloc3DArray( &d_blueArray, &channelDesc, volumeSize ) );

    blueParams.srcPtr   =   make_hipPitchedPtr( blue_map, volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height );
    blueParams.dstArray =   d_blueArray;
    blueParams.extent   =   volumeSize;
    blueParams.kind     =   hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D( &blueParams ) );

    texBlue.normalized = true;
    texBlue.filterMode = hipFilterModeLinear;
    texBlue.addressMode[0] = hipAddressModeClamp;
    texBlue.addressMode[1] = hipAddressModeClamp;
    texBlue.addressMode[2] = hipAddressModeClamp;

    checkCudaErrors( hipBindTextureToArray( texBlue, d_blueArray, channelDesc ) );

    // OUTPUT BUFFER
    checkCudaErrors( hipMalloc( (void**) &d_volume, imageW * imageH * 3 * sizeof(uchar) ) );
    checkCudaErrors( hipMemset( d_volume, 0, imageW * imageH * 3 * sizeof(uchar) ) );
}


extern "C"
void freeCudaBuffers()
{
    checkCudaErrors( hipUnbindTexture(texRed) );
    checkCudaErrors( hipFreeArray(d_redArray) );

    checkCudaErrors( hipUnbindTexture(texGreen) );
    checkCudaErrors( hipFreeArray(d_greenArray) );

    checkCudaErrors( hipUnbindTexture(texBlue) );
    checkCudaErrors( hipFreeArray(d_blueArray) );

    checkCudaErrors( hipFree( d_volume ) );
}


extern "C"
void render_kernel( dim3 gridSize, dim3 blockSize,
                    unsigned char *buffer,
                    uint imageW, uint imageH,
                    float dens, float bright, float offset, float scale )
{
    checkCudaErrors( hipMemset( d_volume, 0, imageW * imageH * 3 * sizeof(unsigned char) ) );
    d_render<<<gridSize,blockSize>>>( d_volume,
                                      imageW, imageH,
                                      dens, bright, offset, scale );
    checkCudaErrors( hipMemcpy( buffer, d_volume, imageW * imageH * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost ) );
}


extern "C"
void copyInvViewMatrix( float *invViewMatrix, size_t sizeofMatrix )
{
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( c_invViewMatrix), invViewMatrix, sizeofMatrix, 0, hipMemcpyHostToDevice ) );
}




